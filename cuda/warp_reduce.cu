#include <cstdio>
#include <hip/hip_runtime.h>


const int WARP_SIZE = 32;

template <typename Tp> __device__ __forceinline__ Tp warp_reduce_sum(Tp val) {
  for (int i = WARP_SIZE >> 1; i >= 1; i >>= 1) {
    val += __shfl_down_sync(0xffffffff, val, i);
  }
  return val;
}

template <typename Tp, const int M, const int N>
__global__ void block_reduce_sum(Tp *dst, Tp *src) {

  constexpr int warp_num = (N + WARP_SIZE - 1) / WARP_SIZE;
  __shared__ Tp data[warp_num];

  int row = blockIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.x;

  int lane_id = col % WARP_SIZE;
  int warp_id = col / WARP_SIZE;

  Tp val = Tp(0);
  if (row < M && col < N) {
    val = src[row * N + col];
  }
  __syncthreads();
  Tp sum = warp_reduce_sum<Tp>(val);
  if (warp_id < warp_num && lane_id == 0) {
    data[warp_id] = sum;
  }
  __syncthreads();
  val = col < warp_num ? data[col] : Tp(0);
  sum = warp_reduce_sum(val);
  __syncthreads();

  if (col == 0) {
    dst[row] = sum;
  }
}

int main() {
  const int M = 32;
  const int N = 1024;

  int *src = static_cast<int *>(malloc(M * N * sizeof(int)));
  int *result = static_cast<int *>(malloc(M * sizeof(int)));
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      src[i * N + j] = i + 1;
    }
  }

  int *d_src, *d_result;
  hipMalloc(&d_src, M * N * sizeof(int));
  hipMalloc(&d_result, M * sizeof(int));

  hipMemcpy(d_src, src, M * N * sizeof(int), hipMemcpyHostToDevice);

  dim3 threads(1, 32);
  int blockX = M;
  int blockY = (N + threads.y - 1) / blockX;
  dim3 block(blockX, blockY);

  block_reduce_sum<int, M, N><<<block, threads>>>(d_result, d_src);

  hipMemcpy(result, d_result, M * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < M; ++i) {
    printf("%d ", result[i]);
  }

  hipFree(d_src);
  hipFree(d_result);
  free(src);
  free(result);

  return 0;
}