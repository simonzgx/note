#include "hip/hip_runtime.h"
#include "reduce.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static const int WARP_SIZE = 32;

template <typename Tp> __device__ __forceinline__ Tp warp_reduce_sum(Tp val) {
#pragma unroll
  for (int i = WARP_SIZE >> 1; i >= 1; i >>= 1) {
    val += __shfl_down_sync(0xffffffff, val, i);
  }
  return val;
}

__device__ void block_reduce_sum(float *dst, float *src, int m, int n) {
  static const int warp_num = 32;
  __shared__ float data[warp_num];

  int row = blockIdx.x;
  int col = blockDim.x * threadIdx.y + threadIdx.x;

  int lane_id = col % WARP_SIZE;
  int warp_id = col / WARP_SIZE;

  float val = row < m && col < n ? src[row * n + col] : float(0);
  __syncthreads();
  float sum = warp_reduce_sum<float>(val);
  if (lane_id == 0) {
    data[warp_id] = sum;
  }
  __syncthreads();
  val = col < warp_num ? data[col] : float(0);
  sum = warp_reduce_sum(val);

  if (col == 0) {
    dst[row] = sum;
  }
}

// in:  M x N
// out: M x 1
// grid(m) block(32, n / 128)
__global__ void block_reduce_sum_vec(float *out, float *in, int m, int n) {
  int row = blockIdx.x;
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  int col = tid * 4;
  int lane_id = tid % WARP_SIZE;
  int warp_id = tid / WARP_SIZE;

  extern __shared__ float data[];
  float4 val = row < m && col < n
                   ? *reinterpret_cast<float4 *>(&in[row * n + col])
                   : make_float4(0, 0, 0, 0);
  float sum = val.x + val.y + val.z + val.w;
  extern __shared__ float data[];
  __syncthreads();
  sum = warp_reduce_sum(sum);
  if (lane_id == 0) {
    data[warp_id] = sum;
  }
  __syncthreads();
  sum = warp_id == 0 ? data[lane_id] : float(0);
  sum = warp_reduce_sum(sum);
  __syncthreads();
  if (warp_id == 0 && lane_id == 0) {
    out[row] = sum;
  }
}

template <typename Tp>
__global__ void reduce_sum(Tp *dst, Tp *src, int m, int n) {
  block_reduce_sum(dst, src, m, n);
}

__global__ void reduce_avg(float *out, float *in, int m, int n) {
  // 先计算sum
  block_reduce_sum(out, in, m, n);
  int row = blockIdx.x;
  if (blockIdx.y == 0 && threadIdx.y == 0) {
    out[row] = out[row] / n;
  }
}

// reduce sum
// grid(m, n/32), block(1, 32)
void g_reduce_sum(float *out, float *in, int m, int n) {
  int blockY = (n + WARP_SIZE - 1) / WARP_SIZE;
  dim3 block(WARP_SIZE, blockY);
  dim3 grid(m);
  reduce_sum<<<grid, block>>>(out, in, m, n);
}

// reduce avg
// grid(m, n/32), block(1, 32)
void g_reduce_avg(float *out, float *in, int m, int n) {
  int blockX = (n + WARP_SIZE - 1) / WARP_SIZE;
  dim3 block(blockX, WARP_SIZE);
  dim3 grid(m);
  reduce_avg<<<grid, block>>>(out, in, m, n);
}

void g_reduce_sum_vec(float *out, float *in, int m, int n) {
  int blockY = (n + WARP_SIZE - 1) / WARP_SIZE;
  blockY = (blockY + 4 - 1) / 4;
  dim3 block(WARP_SIZE, blockY);
  dim3 grid(m);
  int sharedMemSize = blockY;
  printf("blockY:%d sharedMemSize:%d\n", blockY, sharedMemSize);
  block_reduce_sum_vec<<<grid, block, sharedMemSize>>>(out, in, m, n);
}