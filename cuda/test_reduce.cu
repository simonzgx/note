#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "operator/reduce.h"

int main() {
  const int M = 1;
  const int N = 4096;

  float *src = static_cast<float *>(malloc(M * N * sizeof(float)));
  float *result = static_cast<float *>(malloc(M * sizeof(float)));
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      src[i * N + j] = j;
    }
  }

  float *d_src, *d_result;
  hipMalloc(&d_src, M * N * sizeof(float));
  hipMalloc(&d_result, M * sizeof(float));

  hipMemcpy(d_src, src, M * N * sizeof(float), hipMemcpyHostToDevice);

  g_reduce_sum_vec(d_result, d_src, M, N);

  hipMemcpy(result, d_result, M * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < M; ++i) {
    printf("%f ", result[i]);
  }

  hipFree(d_src);
  hipFree(d_result);
  free(src);
  free(result);

  return 0;
}